#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include "routinesGPU.h"
#include "routinesCPU.h"

// Thread block size
#define BLOCK_SIZE 16 

// Globals
const float PI = 3.141593;
const float level = 1000.0f;

// Forward declarations of functions
__global__ void noise_reduction(uint8_t*, float*, int, int);
__global__ void img_gradient(float*,float*, float*,float*, float*, int, int);
__global__ void no_max_supression(uint8_t *,uint8_t * ,float* , float* , int ,int , float, float);
__global__ void houghtransform(uint8_t *, int , int ,float ,float , float , uint32_t *, int , int , float *, float *);

void getlines(int , uint32_t *, int , int , int , int , float *, float *,int *, int *, int *, int *, int *);

void line_asist_GPU(uint8_t *im, int height, int width, float *sin_table, float *cos_table, uint32_t *accum, int *x1, int *x2, int *y1, int *y2, int *nlines) {

	//Noise Reduction
	float* NR_d;
	uint8_t* img_in_d;
	int sizeNR = height * width * sizeof(float);
	int sizeIMG = sizeof(uint8_t) * width * height;

	hipMalloc((void**)&NR_d, sizeNR);
	hipMalloc((void**)&img_in_d, sizeIMG);
	hipMemcpy(img_in_d, im, size, hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(width / dimBlock.x, height / dimBlock.y);

	noise_reduction<<<dimGrid,dimBlock>>>(img_in_d,NR_d,height,width);

	//Img Gradient
	float* Gx_d,*Gy_d,*G_d,*phi_d;
	int sizeXY = height * width * sizeof(float);

	hipMalloc((void**)&Gx_d, sizeXY);	
	hipMalloc((void**)&Gy_d, sizeXY);
	hipMalloc((void**)&G_d, sizeXY);							
	hipMalloc((void**)&phi_d, sizeXY);	

	img_gradient<<<dimGrid,dimBlock>>>(NR_d,Gx_d,Gy_d,G_d,phi_d,height,width);

	//No max Supresion
	uint8_t* imgEdge_d, *pedge_d;
	int sizePE = sizeof(uint8_t) * width * height;
	hipMalloc((void**)&imgEdge_d, sizePE);
	hipMalloc((void**)&pedge_d, sizePE);

	float lowthres = level/2;
	float hithres  = 2*(level);

	no_max_supression<<<dimGrid,dimBlock>>>(pedge_d,imgEdge_d,G_d,phi_d,height,width,lowthres,hithres);

	//Hough Transform
	float* sin_table_d, * cos_table_d;
	int sizeTABLE = 180 * sizeof(float);
	hipMalloc((void**)&sin_table_d, sizeTABLE);
	hipMalloc((void**)&cos_table_d, sizeTABLE);
	hipMemcpy(sin_table_d, sin_table, sizeTABLE, hipMemcpyHostToDevice);
	hipMemcpy(cos_table_d, cos_table, sizeTABLE, hipMemcpyHostToDevice);

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);
	float center_x = width/2.0; 
	float center_y = height/2.0;
	int accu_height = hough_h * 2.0;
	int accu_width  = 180;

	uint32_t *accum_d;
	int sizeACCUM = accu_width*accu_height * sizeof(uint32_t);																
	hipMalloc((void**)&accum_d, sizeACCUM);
	hipMemset(&accum_d, 0, sizeACCUM);

	houghtransform<<<dimGrid,dimBlock>>>(imgEdge_d,width, height,hough_h, center_x,center_y,accum_d,accu_width,accu_height,sin_table_d,cos_table_d);

	int threshold = height/6;
	if (width>height) threshold = width/6;
	hipMemcpy(accum, accum_d, size, hipMemcpyDeviceToHost);

	//Get Lines
	getlines(threshold, accum, accu_width, accu_height, width, height, sin_table, cos_table, x1, y1, x2, y2, nlines);

	// Free device memory
	hipFree(NR_d);
	hipFree(img_in_d);
	hipFree(Gx_d);
	hipFree(Gy_d);
	hipFree(G_d);
	hipFree(phi_d);
	hipFree(imgEdge_d);
	hipFree(pedge_d);
	hipFree(accum_d);
	hipFree(cos_table_d);
	hipFree(sin_table_d);
}

__global__ void noise_reduction(uint8_t* im,float* NR, int height, int width) {

	int i = blockIdx.y * blockDim.y + threadIdx.y + 2;						
	int j = blockIdx.x * blockDim.x + threadIdx.x + 2;						

	if(i < height-2 && j < width-2) {
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;

	}
}

__global__ void img_gradient(float* NR,float* Gx, float* Gy,float* G, float* phi, int height, int width) {

	int i = blockIdx.y * blockDim.y + threadIdx.y + 2;						
	int j = blockIdx.x * blockDim.x + threadIdx.x + 2;						

	if(i < height-2 && j < width-2) {
		Gx[i*width+j] = 
				(1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
			+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
			+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
			+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
			+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


		Gy[i*width+j] = 
				((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
			+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
			+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
			+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);


		// CALCULO DE G Y DE PHI	(G = √Gx²+Gy²)
		G[i*width+j] = sqrtf((Gx[i*width+j]*Gx[i*width+j])+(Gy[i*width+j]*Gy[i*width+j]));				
		phi[i*width+j] = atan2f(fabs(Gy[i*width+j]),fabs(Gx[i*width+j]));

		if(fabs(phi[i*width+j])<=PI/8 ){
			phi[i*width+j] = 0;
		}
		else if (fabs(phi[i*width+j])<= 3*(PI/8)){
			phi[i*width+j] = 45;
		}
		else if (fabs(phi[i*width+j]) <= 5*(PI/8)){
			phi[i*width+j] = 90;
		}
		else if (fabs(phi[i*width+j]) <= 7*(PI/8)){
			phi[i*width+j] = 135;
		}
		else{
			phi[i*width+j] = 0;
		}
	}
}

__global__ void no_max_supression(uint8_t *pedge,uint8_t *img_out ,float* G, float* phi, int height,int width, float lowthres, float hithres ) {

	int i = blockIdx.y * blockDim.y + threadIdx.y + 3;						
	int j = blockIdx.x * blockDim.x + threadIdx.x + 3;						

	if(i < height-3 && j < width-3) {

		pedge[i*width+j] = 0;
		if(phi[i*width+j] == 0){
			if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]){
				pedge[i*width+j] = 1;
			} 
		} 
		else if(phi[i*width+j] == 45) {
			if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]){
				pedge[i*width+j] = 1;
			}
		} 
		else if(phi[i*width+j] == 90) {
			if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]){
				pedge[i*width+j] = 1;
			}
		} 
		else if(phi[i*width+j] == 135) {
			if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]){
				pedge[i*width+j] = 1;
			}		
		}
	}

	int ii,jj;

	if(i < height-3 && j < width-3) {
		img_out[i*width+j] = 0;
		if(G[i*width+j]>hithres && pedge[i*width+j]){
			img_out[i*width+j] = 255;
		}
		else if(pedge[i*width+j] && G[i*width+j]>=lowthres && G[i*width+j]<hithres){
			for (ii=-1;ii<=1; ii++){
				for (jj=-1;jj<=1; jj++){
					if (G[(i+ii)*width+j+jj]>hithres){
						img_out[i*width+j] = 255;
					}
				} 
			} 
		}				
	}
}

__global__ void houghtransform(uint8_t *im, int width, int height,float hough_h,float center_x, float center_y, uint32_t *accumulators, int accu_width, int accu_height,  float *sin_table, float *cos_table) {

	int i = blockIdx.y * blockDim.y + threadIdx.y;						
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if(i < height && j < width){
		if(im[ (i*width) + j] > 250){						
			for(int theta=0;theta<180;theta++){  
				float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
				atomicAdd(&accumulators[ (int)((round(rho + hough_h) * 180.0)) + theta], 1);
			} 
		} 
	}
}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, float *sin_table, float *cos_table, int *x1, int *y1, int *x2, int *y2, int *lines)
{
	int rho, theta;
	uint32_t max;

	for(rho=0;rho<accu_height;rho++){
		for(theta=0;theta<accu_width;theta++){  
			if(accumulators[(rho*accu_width) + theta] >= threshold){  
				
				max = accumulators[(rho*accu_width) + theta]; 
				for(int ii=-4;ii<=4;ii++){  
					for(int jj=-4;jj<=4;jj++){  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) ){  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max ){
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]){
					int x1_aux, y1_aux, x2_aux, y2_aux;  
					x1_aux = y1_aux = x2_aux = y2_aux = 0;  

					if(theta >= 45 && theta <= 135){
						if (theta > 90){
							x1_aux = width/2;  
							y1_aux = ((float)(rho-(accu_height/2)) - ((x1_aux - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2_aux = width;  
							y2_aux = ((float)(rho-(accu_height/2)) - ((x2_aux - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} 
						else{  
							x1_aux = 0;  
							y1_aux = ((float)(rho-(accu_height/2)) - ((x1_aux - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2_aux = width*2/5;  
							y2_aux = ((float)(rho-(accu_height/2)) - ((x2_aux - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} 
					else{  
						y1_aux = 0;  
						x1_aux = ((float)(rho-(accu_height/2)) - ((y1_aux - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2_aux = height;  
						x2_aux = ((float)(rho-(accu_height/2)) - ((y2_aux - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}

					x1[*lines] = x1_aux;
					y1[*lines] = y1_aux;
					x2[*lines] = x2_aux;
					y2[*lines] = y2_aux;
					(*lines)++;
				}
			}
		}
	}
}